
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define DIM 2048
#define N (DIM*DIM)
#define THREAD_PER_BLOCK 512

__global__ void add(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void randomInts(int* a, int size)
{
    std::random_device random;
    for (int i = 0 ; i < size ; ++i)
        a[i] = random() % 1000;
}

void printMatrix(int* m, int dim)
{
    printf("matrix %p :\n", m);
    for(int i = 0 ; i < 3 ; ++i)
    {
        for(int j = 0 ; j < 2 ; ++j)
            printf("%d\t", m[i+j*dim]);
        printf("%d...\n", m[i+2*dim]);
    }
    printf("...\n\n");
}

// nvcc -o bin/exo2 src/exo2.cu
int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    randomInts(a, N);
    b = (int*)malloc(size);
    randomInts(b, N);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    auto t1 = std::chrono::high_resolution_clock::now();

    add<<<N/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(d_a, d_b, d_c);

    auto t2 = std::chrono::high_resolution_clock::now();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche les 3 premières lignes/col de la mat1, mat2
    // et mat résultat
    printMatrix(a, DIM);
    printMatrix(b, DIM);
    printMatrix(c, DIM);

    auto int_us = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    std::cout<<"done in "<<int_us.count()<<" μs"<<std::endl;

    free(a);
    free(b);
    free(c);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    return 0;
}