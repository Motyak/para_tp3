
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define NB_OF_ELEM 16777216
#define DIM 4096
#define MAX_NB_THREADS 1024
//nb de mesures pour le calcul du temps moyen d'execution
#define SAMPLE_SIZE 10

__global__ void multiply(int* a, int* b, int* c)
{
    // // je calcule le nombre de threads dans la grid,
    // // sa racine carré correspond a la dim des matrices
    // int dim = sqrtf(gridDim.x * blockDim.x);

    int index_c = threadIdx.x + blockIdx.x * blockDim.x;
    int index_a = ((int)(index_c / DIM)) * DIM;
    int index_b = index_c % DIM;

    c[index_c] = 0;
    for(int i = 0 ; i < DIM ; ++i)
    {
        c[index_c] += a[index_a] * b[index_b];
        ++index_a;      //pas de 1
        index_b += DIM; //pas de dim
    }
}

void randomInts(int* a, int size)
{
    std::random_device random;
    for (int i = 0 ; i < size ; ++i)
        a[i] = random() % 1000;
}

void printMatrix(int* m, int dim)
{
    printf("matrix %p :\n", m);
    for(int i = 0 ; i < 3 ; ++i)
    {
        for(int j = 0 ; j < 2 ; ++j)
            printf("%d\t", m[i+j*dim]);
        printf("%d...\n", m[i+2*dim]);
    }
    printf("...\n\n");
}

void resultTest()
{
    printf("DEBUT TEST RESULTAT\n");

    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NB_OF_ELEM * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size); randomInts(a, NB_OF_ELEM);
    b = (int*)malloc(size); randomInts(b, NB_OF_ELEM);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    multiply<<<NB_OF_ELEM/MAX_NB_THREADS,MAX_NB_THREADS>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche les 3 premières lignes/col de la mat1, mat2
    // et mat résultat
    printMatrix(a, DIM);    printMatrix(b, DIM);    printMatrix(c, DIM);

    free(a);        free(b);        free(c);
    hipFree(d_a);  hipFree(d_b);  hipFree(d_c);

    printf("FIN TEST RESULTAT\n\n");
}

void speedTest(int nbOfBlocks, int nbThreadsPerBlock)
{
    printf("%d BLOCS ET %d THREADS/BLOC\n", nbOfBlocks, nbThreadsPerBlock);

    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NB_OF_ELEM * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size); 
    b = (int*)malloc(size); 
    c = (int*)malloc(size);

    int t_ns = 0;

    for(int i = 1 ; i <= SAMPLE_SIZE ; ++i)
    {
        randomInts(a, NB_OF_ELEM);
        randomInts(b, NB_OF_ELEM);
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        auto t1 = std::chrono::high_resolution_clock::now();
        multiply<<<nbOfBlocks,nbThreadsPerBlock>>>(d_a, d_b, d_c);
        auto t2 = std::chrono::high_resolution_clock::now();
        std::cout<<std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()<<std::endl;
        t_ns += std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count();
    }

    std::cout<<"done in "<<t_ns / SAMPLE_SIZE <<" ns (in average)"<<std::endl<<std::endl;

    free(a);        free(b);        free(c);
    hipFree(d_a);  hipFree(d_b);  hipFree(d_c);
}

// nvcc -o bin/exo3 src/exo3.cu
int main(void)
{
    resultTest();

    printf("DEBUT TESTS DE VITESSE\n");
    speedTest(NB_OF_ELEM, 1);
    speedTest(524288, 32);
    speedTest(NB_OF_ELEM/MAX_NB_THREADS, MAX_NB_THREADS);
    printf("FIN TESTS DE VITESSE\n");

    return 0;
}