
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define DIM 2048
#define N (DIM*DIM)
#define THREAD_PER_BLOCK 512

__global__ void multiply(int* a, int* b, int* c)
{
    // // je calcule le nombre de threads dans la grid,
    // // sa racine carré correspond a la dim des matrices
    // int dim = sqrtf(gridDim.x * blockDim.x);

    int index_c = threadIdx.x + blockIdx.x * blockDim.x;
    int index_a = ((int)(index_c / DIM)) * DIM;
    int index_b = index_c % DIM;

    c[index_c] = 0;
    for(int i = 0 ; i < DIM ; ++i)
    {
        c[index_c] += a[index_a] * b[index_b];
        ++index_a;      //pas de 1
        index_b += DIM; //pas de dim
    }
}

void randomInts(int* a, int size)
{
    std::random_device random;
    for (int i = 0 ; i < size ; ++i)
        a[i] = random() % 1000;
}

void printMatrix(int* m, int dim)
{
    printf("matrix %p :\n", m);
    for(int i = 0 ; i < 3 ; ++i)
    {
        for(int j = 0 ; j < 2 ; ++j)
            printf("%d\t", m[i+j*dim]);
        printf("%d...\n", m[i+2*dim]);
    }
    printf("...\n\n");
}

// nvcc -o bin/exo3 src/exo3.cu
int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    randomInts(a, N);
    b = (int*)malloc(size);
    randomInts(b, N);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    auto t1 = std::chrono::high_resolution_clock::now();

    multiply<<<N/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(d_a, d_b, d_c);

    auto t2 = std::chrono::high_resolution_clock::now();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche les 3 premières lignes/col de la mat1, mat2
    // et mat résultat
    printMatrix(a, DIM);
    printMatrix(b, DIM);
    printMatrix(c, DIM);

    auto int_us = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    std::cout<<"done in "<<int_us.count()<<" μs"<<std::endl;

    free(a);
    free(b);
    free(c);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    return 0;
}