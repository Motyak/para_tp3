
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define NB_OF_ELEM 16777216
#define MAX_NB_THREADS 1024
//nb de mesures pour le calcul du temps moyen d'execution
#define SAMPLE_SIZE 10

__global__ void add(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void randomInts(int* a, int nbOfElem)
{
    std::random_device random;
    for (int i = 0 ; i < nbOfElem ; ++i)
        a[i] = random() % 1000;
}

void resultTest()
{
    printf("DEBUT TEST RESULTAT\n");

    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NB_OF_ELEM * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size); randomInts(a, NB_OF_ELEM);
    b = (int*)malloc(size); randomInts(b, NB_OF_ELEM);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<NB_OF_ELEM/MAX_NB_THREADS,MAX_NB_THREADS>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche que les 10 premieres pour verifier
    for(int i = 0 ; i < 10 ; ++i)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    free(a);        free(b);        free(c);
    hipFree(d_a);  hipFree(d_b);  hipFree(d_c);

    printf("FIN TEST RESULTAT\n\n");
}

void speedTest(int nbOfBlocks, int nbThreadsPerBlock)
{
    printf("%d BLOCS ET %d THREADS/BLOC\n", nbOfBlocks, nbThreadsPerBlock);

    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NB_OF_ELEM * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size); 
    b = (int*)malloc(size); 
    c = (int*)malloc(size);

    int t_ns = 0;

    for(int i = 1 ; i <= SAMPLE_SIZE ; ++i)
    {
        randomInts(a, NB_OF_ELEM);
        randomInts(b, NB_OF_ELEM);
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        auto t1 = std::chrono::high_resolution_clock::now();
        add<<<nbOfBlocks,nbThreadsPerBlock>>>(d_a, d_b, d_c);
        auto t2 = std::chrono::high_resolution_clock::now();
        std::cout<<std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()<<std::endl;
        t_ns += std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count();
    }

    std::cout<<"done in "<<t_ns / SAMPLE_SIZE <<" ns (in average)"<<std::endl<<std::endl;

    free(a);        free(b);        free(c);
    hipFree(d_a);  hipFree(d_b);  hipFree(d_c);
}

// nvcc -o bin/exo1 src/exo1.cu
int main(void)
{
    resultTest();

    printf("DEBUT TESTS DE VITESSE\n");
    speedTest(NB_OF_ELEM, 1);
    speedTest(524288, 32);
    speedTest(NB_OF_ELEM/MAX_NB_THREADS, MAX_NB_THREADS);
    printf("FIN TESTS DE VITESSE\n");

    return 0;
}