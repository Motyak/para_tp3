
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define NB_OF_ELEM 16777216
#define MAX_NB_THREADS 1024

__global__ void add(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void randomInts(int* a, int nbOfElem)
{
    std::random_device random;
    for (int i = 0 ; i < nbOfElem ; ++i)
        a[i] = random() % 1000;
}

// nvcc -o bin/exo1 src/exo1.cu
int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NB_OF_ELEM * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    randomInts(a, NB_OF_ELEM);
    b = (int*)malloc(size);
    randomInts(b, NB_OF_ELEM);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    auto t1 = std::chrono::high_resolution_clock::now();

    // add<<<NB_OF_ELEM,1>>>(d_a, d_b, d_c);    //avg: 40000ns
    // add<<<524288,32>>>(d_a, d_b, d_c);       //avg: 42000ns
    add<<<16384,1024>>>(d_a, d_b, d_c);      //avg: 38000ns

    auto t2 = std::chrono::high_resolution_clock::now();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche que les 10 premieres pour verifier
    for(int i = 0 ; i < 10 ; ++i)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    auto int_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1);
    std::cout<<"done in "<<int_ns.count()<<" ns"<<std::endl;

    free(a);
    free(b);
    free(c);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    return 0;
}