
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <random>

#define N 512

__global__ void add(int* a, int* b, int* c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void randomInts(int* a, int size)
{
    std::random_device random;
    for (int i = 0 ; i < size ; ++i)
        a[i] = random() % 1000;
}

// nvcc -o bin/exo1 src/exo1.cu
int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    randomInts(a, N);
    b = (int*)malloc(size);
    randomInts(b, N);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0 ; i < N ; ++i)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    free(a);
    free(b);
    free(c);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    return 0;
}