
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>

#include <iostream>
#include <random>
#include <chrono>

#define N  (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void add(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void randomInts(int* a, int size)
{
    std::random_device random;
    for (int i = 0 ; i < size ; ++i)
        a[i] = random() % 1000;
}

// nvcc -o bin/exo1 src/exo1.cu
int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    randomInts(a, N);
    b = (int*)malloc(size);
    randomInts(b, N);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    auto t1 = std::chrono::high_resolution_clock::now();

    add<<<N/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(d_a, d_b, d_c);

    auto t2 = std::chrono::high_resolution_clock::now();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // j'affiche que les 10 premieres pour verifier
    for(int i = 0 ; i < 10 ; ++i)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    auto int_us = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    std::cout<<"done in "<<int_us.count()<<" μs"<<std::endl;

    free(a);
    free(b);
    free(c);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    return 0;
}