
#include <hip/hip_runtime.h>
// FAIRE LA MEME CHOSE QUE EXO1,
// TOUT DANS UN TABLEAU UNIDIMENSIONNEL
// POUR QUE LES DONNES SOIENT CONTINGENTES DANS
// LA MEMOIRE (POUR LE MEMCOPY HOST<=>DEVICE
// https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf

// #include <stdio.h>
// // #include <cuda.h>

// #include <iostream>
// #include <random>
// #include <chrono>

// #define DIM 2048

// __global__ void add(int** a, int** b, int** c)
// {
//     int& i = blockIdx.x;
//     int& j = threadIdx.x;
//     c[i][j] = a[i][j] + b[i][j];
// }

// void randomInts(int** a, int dim)
// {
//     std::random_device random;
//     for (int i = 0 ; i < dim ; ++i)
//         for (int j = 0 ; j < dim ; ++j)
//             a[i][j] = random() % 1000;
// }

// // nvcc -o bin/exo1 src/exo1.cu
// int main(void)
// {
//     int **a, **b, **c;
//     int **d_a, **d_b, **d_c;
//     int size = DIM * sizeof(int);

//     cudaMalloc((void**)&d_a, size);
//     for(int i = 0 ; i < DIM ; ++i)
//         cudaMalloc((void**)&d_a[i], size);

//     cudaMalloc((void**)&d_b, size);
//     for(int i = 0 ; i < DIM ; ++i)
//         cudaMalloc((void**)&d_b[i], size);

//     cudaMalloc((void**)&d_c, size);
//     for(int i = 0 ; i < DIM ; ++i)
//         cudaMalloc((void**)&d_c[i], size);

//     a = (int*)malloc(size);
//     for(int i = 0 ; i < DIM ; ++i)
//         a[i] = (int*)malloc(size);
//     randomInts(a, DIM);
//     b = (int*)malloc(size);
//     for(int i = 0 ; i < size ; ++i)
//         b[i] = (int*)malloc(size);
//     randomInts(b, DIM);
//     c = (int*)malloc(size);
//     for(int i = 0 ; i < DIM ; ++i)
//         c[i] = (int*)malloc(size);

//     cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
//     cudaMemcpy(d_b, b, size, cudaMemcpyHostToDevice);

//     auto t1 = std::chrono::high_resolution_clock::now();

//     add<<<N/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(d_a, d_b, d_c);

//     auto t2 = std::chrono::high_resolution_clock::now();

//     cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

//     // j'affiche que les 10 premieres pour verifier
//     for(int i = 0 ; i < 10 ; ++i)
//         printf("%d + %d = %d\n", a[i], b[i], c[i]);

//     auto int_us = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
//     std::cout<<"done in "<<int_us.count()<<" μs"<<std::endl;

//     free(a);
//     free(b);
//     free(c);
//     cudaFree(d_a); 
//     cudaFree(d_b); 
//     cudaFree(d_c);

//     return 0;
// }